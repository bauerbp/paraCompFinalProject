#include "hip/hip_runtime.h"
#define NX 8
#define BATCH_SIZE 1

#include "hipfft/hipfft.h"

#include <math.h>
#include <stdio.h>

//#include "soundfile-2.2/libsoundfile.h"
typedef float2 Complex;

void testcuFFT(){

  hipfftReal *h_signal = (hipfftReal *)malloc(sizeof(hipfftReal) * BATCH_SIZE);
  hipfftComplex *h_data = (hipfftComplex *)malloc(sizeof(hipfftComplex) * (NX/2+1)*BATCH_SIZE);

  float ryanSignal [NX] = {0.0,
1.15443278102,
1.50377819535,
0.957393116649,
0.19925316202,
0.0408603874003,
0.663651234058,
1.44858683588};
  // Initalize the memory for the signal
  for (unsigned int i = 0; i < NX; ++i)
  {
    //h_signal[i].x = rand() / (float)RAND_MAX+1;
    h_signal[i] = ryanSignal[i];
    //h_signal[i].y = 0;
    printf("h_signal[%u]: %f\n", i, h_signal[i]);
  }

  hipfftHandle plan;
  hipfftComplex *d_data;
  hipfftReal *d_signal;
  hipMalloc((void**)&d_data, sizeof(hipfftComplex)*(NX/2+1)*BATCH_SIZE);
  hipMalloc((void**)&d_signal, sizeof(hipfftReal)*NX);
  //hipMalloc((void**)&d_signal, sizeof(hipfftReal)*SIGNAL_SIZE);
  hipMemcpy(d_signal, h_signal, sizeof(hipfftReal)*NX, hipMemcpyHostToDevice);

  free(h_signal);

  if(hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return; 
  }
  if(hipfftPlan1d(&plan, NX, HIPFFT_R2C, BATCH_SIZE) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    return; 
  } 

  // Use the CUFFT plan to transform the signal in place. 
  if(hipfftExecR2C(plan, (hipfftReal*)d_signal, d_data) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
    return; 
  }
  if(hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return; 
  }
  
  hipMemcpy(h_data,d_data,sizeof(hipfftComplex)*BATCH_SIZE * (NX/2+1),hipMemcpyDeviceToHost);
  for(unsigned int k=0; k<10; k++){
    //printf("h_data[%i]: %f\n",k,h_data[k].x);
    printf("h_data[%u]: %f\n", k, h_data[k].x);
  }

  hipfftDestroy(plan);
  hipFree(d_data);
}

int main(){
  testcuFFT(); 

  return 0;
}
